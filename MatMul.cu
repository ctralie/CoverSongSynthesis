
#include <hip/hip_runtime.h>
/*Programmer: Chris Tralie
Purpose: To create fast 2D convolutional matrix multiplication code
as parallel CUDA kernels
*/

__global__ void MatMulNaive(float* A, float* B, float* C, int M, int K, int N) {
    /*
    A: MxK matrix
    B: KxN matrix
    C: MxN matrix
    */
    int i = blockIdx.x;
    int j = threadIdx.x;
    int k = 0;
    float res = 0.0;
    for (k = 0; k < K; k++) {
        res += A[i*K + k]*B[k*N+j];
    }
    C[i*N+j] = res;
}


__global__ void MatMulConv2D(float* W, float* H, float* Lam, int M, int N, int K, 
        int T, int F, int TBlocks, int FBlocks) {
    /*
    Perform 2D convolutional matrix multiplication
    :param W: An MxKxT input matrix
    :param H: A KxNxF input matrix
    :param Lam: A MxN output matrix
    :param M, N, K, T, F: Dimensions
    :param TBlocks: Number of blocks of T padding to load in per grid block
    :param FBlocks: Number of blocks of F padding to load in per grid block
    */

    /*Shared Memory Layout in x, which holds chunks of W and H that are 
        shared with overlapping convolutions.  For a block size of B:
            1) W goes from i-F:i+B-1, k, 0:T-1.
            2) H goes from k, j-T:j+B-1, 0:F at an offset of (F+B)*T
    */
    extern __shared__ float x[]; 
    int hoff = (F+blockDim.x)*T; //Offset of H chunk in shared memory
    //TODO: Think about row major coalescing with order of access
    int iblock = blockIdx.x*blockDim.x;
    int jblock = blockIdx.y*blockDim.y;
    int i = iblock + threadIdx.x;
    int j = jblock + threadIdx.y;
    int KT = K*T;
    int NF = N*F;
    int k, t, f;
    int thist, thisf;
    int thisi, thisj;
    float res = 0.0;
    //Loop over all K separately
    for (k = 0; k < K; k++) {
        //Step 1: Load chunks of W into shared memory
        //W goes from iblock-F+1:iblock+B-1, k, 0:T-1
        for (f = 0; f < FBlocks+1; f++) {
            if (f == FBlocks) {
                //On the last one, copy over interval from [iblock, iblock+B-1]
                thisi = i; 
                thisf = F+threadIdx.x;
            }
            else{ 
                //For the other chunks, copy over interval from [iblock-F, iblock-1]
                thisi = i-F+f*blockDim.x;
                if (thisi >= iblock) {
                    continue; //Past F boundary for block at iblock-1
                }
                thisf = f*blockDim.x+threadIdx.x;
            }
            for (t = 0; t < TBlocks; t++) {
                thist = t*blockDim.y + threadIdx.y;
                if (thist >= T) {
                    continue;
                }
                //Pull out W[thisi, k, thist]
                if (thisi < 0 || thisi >= M) {
                    x[T*thisf+thist] = 0;
                }
                else {
                    x[T*thisf+thist] = W[thisi*KT+k*T+thist];
                }
            }
        }
        __syncthreads();
        //Step 2: Load chunks of H into shared memory
        //H goes from k, jblock-T+1:jblock+B-1, 0:F at an offset of (F+B)*T
        for (t = 0; t < TBlocks+1; t++) {
            if (t == TBlocks) {
                //On the last one, copy over interval from [jblock:jblock+B-1]
                thisj = j; 
                thist = T+threadIdx.y;
            }
            else {
                //For the other chunks, copy over interval from [jblock-T:jblock-1]
                thisj = j-T+t*blockDim.y;
                if (thisj >= jblock) {
                    continue; //Past T boundary for block at jblock-1
                }
                thist = t*blockDim.y+threadIdx.y;
            }
            for (f = 0; f < FBlocks; f++) {
                thisf = f*blockDim.x + threadIdx.x;
                if (thisf >= F) {
                    continue;
                }
                //Pull out H[k, thisj, f] and put in at an offset
                if (thisj < 0 || thisj >= N) {
                    x[hoff + F*thist+thisf] = 0;
                }
                else{
                    x[hoff + F*thist+thisf] = H[k*NF + thisj*F + thisf];
                }
            }
        }
        __syncthreads();

        //Step 3: Do matrix multiplication
        for (f = 0; f < F; f++) {
            for (t = 0; t < T; t++) {
                //W[i-f, k, t]*H[k, j-t, f]
                res += x[(F+threadIdx.x-f)*T + t]*x[hoff+(T+threadIdx.y-t)*F+f];
            }
        }
        __syncthreads();//The lack of this sync at the end of each k
        // was causing a major bug!!!
    }
    if (i < M && j < N) {
        Lam[i*N+j] = res;
    }
}


__global__ void MatMulConv2DWGrad(float* W, float* H, float* V, float* VLam, 
        int M, int N, int K, int T, int F, int FBlocks) {
    /*
    Perform 2D convolutional matrix multiplicative update of W
    :param W: An MxKxT input matrix
    :param H: A KxNxF input matrix
    :param V: A MxN target matrix
    :param VLam: An MxN matrix which is W*H 2DConv
    :param VLam: The current MxN approximation of V
    :param M, N, K, T, F: Dimensions
    :param FBlocks: Number of blocks of F padding to load in per grid block
    */

    /*Shared Memory Layout in x, which holds chunks of V/VLam and H that are 
        shared with overlapping convolutions.  For a block size of B:
            1) V goes from iblock:iblock+B+F-1, j:j+B-1
            2) VLam goes from iblock:iblock+B+F-1, j:j+B-1 at an offset of
                (B+F)*B
            3) H goes from k, j-T:j+B-1, 0:F at an offset of 2*(B+F)*B

            NOTE: Assuming 1 block of t values
    */
    extern __shared__ float x[]; 
    int vlamoff = (blockDim.x+F)*blockDim.x; //Offset of VLam chunk in shared memory
    int hoff = 2*vlamoff; //Offset of H chunk in shared memory
    int iblock = blockIdx.x*blockDim.x;
    int k = blockIdx.y;
    int tblock = blockIdx.z*blockDim.z;
    int jblock;
    int i = iblock + threadIdx.x;
    int t = tblock + threadIdx.y;
    int NF = N*F;
    int j, f;
    int thist, thisf;
    int thisi, thisj;
    float num = 0.0;
    float denom = 0.0;

    //Loop over chunks of the j dimension
    for (jblock = 0; jblock < N; jblock += blockDim.x) {
        j = jblock+threadIdx.y;
        //Step 1: Copy out sections of V and VLam
        for (f = 0; f < FBlocks+1; f++) {
            if (f == FBlocks) {
                //On the last one, copy over interval from [iblock, iblock+B-1]
                thisi = i; 
                thisf = threadIdx.x;
            }
            else{ 
                //For the other chunks, copy over interval from [iblock+B, iblock+B+F-1]
                thisf = blockDim.x+f*blockDim.x+threadIdx.x;
                if (thisf >= blockDim.x+F) {
                    continue; //Past F boundary
                }
                thisi = i+(f+1)*blockDim.x;
            }
            //Pull out V[thisi, j] and VLam[thisi, j]
            if (thisi < 0 || thisi >= M || j >= N) {
                x[blockDim.x*thisf+threadIdx.y] = 0;
                x[vlamoff+blockDim.x*thisf+threadIdx.y] = 0;
            }
            else {
                x[blockDim.x*thisf+threadIdx.y] = V[thisi*N+j];
                x[vlamoff+blockDim.x*thisf+threadIdx.y] = VLam[thisi*N+j];
            }

        }
        __syncthreads();
        //Step 2: Copy out sections of H
        //H goes from k, j-T:j+B-1, 0:F
        /*if (threadIdx.y <= T) {
            //Copy out [jblock-T, jblock] section
            thisj = jblock - threadIdx.y;
            thist = T - threadIdx.y;
            for (f = 0; f < F; f++) {
                if (thisj < 0 || thisj >= M) {
                    x[hoff+thist*F+f] = 0;
                }
                else {
                    x[hoff+thist*F+f] = H[k*NF + thisj*F + f];
                }
            }
        }
        //Copy out [1, blockdim-1] part
        if (threadIdx.y > 0) {
            thisj = j;
            thist = T + threadIdx.y;
            for (f = 0; f < F; f++) {
                if (thisj >= M) {
                    x[hoff+thist*F+f] = 0;
                }
                else{
                    x[hoff+thist*F+f] = H[k*NF + thisj*F + f];
                }
            }
        }
        __syncthreads();
        */


        //Step 3: Do multiplication
        for (f = 0; f < F; f++) {
            for (thisj = 0; thisj < blockDim.y; thisj++) {
                //V[i+f, j] * H[k, j-t, f]
                //num += x[(threadIdx.x+f)*T+thisj]*x[hoff+(blockDim.y-threadIdx.y+thisj)*F+f];
                num += x[(threadIdx.x+f)*blockDim.x+thisj];
            }
        }
        __syncthreads();
    }
    if (i < M && t < T) {
        //W[i, k, t]
        W[i*K*T + k*T + t] = num;
    }
}